#include "hip/hip_runtime.h"
/*
    Ensures safe cuda application executions
*/
#define gpuSafeExec(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    Clears shared memory which is not full of previous
    numbers. Shmem is remembers values between consecutive
    kernel calls.
*/
__device__ void flushShmem(float *shmem, int shmemSize){
    for (int i = 0; i < shmemSize; i ++)
        shmem[i] = 0.0f;
    return;
}

// 12 bytes, should avoid bank conflicts*
struct sPoint{
    float x;
    float y;
    float z;
};

__device__ float diff(float Axi, float Axj, float Ayi, float Ayj, float Azi, float Azj
, float Bxi, float Bxj, float Byi, float Byj, float Bzi, float Bzj){
        float da = sqrt((Axi-Axj)*(Axi-Axj)
                    + (Ayi-Ayj)*(Ayi-Ayj)
                    + (Azi-Azj)*(Azi-Azj));
        float db = sqrt((Bxi-Bxj)*(Bxi-Bxj)
                    + (Byi-Byj)*(Byi-Byj)
                    + (Bzi-Bzj)*(Bzi-Bzj));
        return (da-db) * (da-db);
} 

/*
Solved by 1d array reduction described by NVIDIA docs.
Might be improved with 2d array reduction?
*/
const int blocksize = 256;
__global__ void galaxy_similarity_reduction(const sGalaxy A, const sGalaxy B, const int n , float* output) {
    __shared__ float sdata[blocksize];
    __shared__ float3 As[blocksize];
    __shared__ float3 Bs[blocksize];

    unsigned int tx_g = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;
    unsigned int bx = blockIdx.x;

    //clear SHMEM
    if (tx == 0)
    {
        flushShmem(sdata, blocksize);
    }

    //wait for shem flush
    __syncthreads();

    for (int tile = 0; tile < n / blocksize; tile++)
    {
        As[tx].x = A.x[tile*blocksize+tx];
        As[tx].y = A.y[tile*blocksize+tx];
        As[tx].z = A.z[tile*blocksize+tx];
        Bs[tx].x = B.x[tile*blocksize+tx];
        Bs[tx].y = B.y[tile*blocksize+tx];
        Bs[tx].z = B.z[tile*blocksize+tx];

        float Ax = A.x[tile*blocksize+tx];
        float Ay = A.y[tile*blocksize+tx];
        float Az = A.z[tile*blocksize+tx];
        float Bx = B.x[tile*blocksize+tx];
        float By = B.y[tile*blocksize+tx];
        float Bz = B.z[tile*blocksize+tx];
        float tmp = 0.0f;
        __syncthreads();
        for (int j = 1; j < blocksize; j++){
            int idx = j + (blocksize * tile); //global index   
            if (idx < tx_g || idx == tx_g){continue;}
            
            float da = sqrt((Ax-As[j].x)*(Ax-As[j].x)
                        + (Ay-As[j].y)*(Ay-As[j].y)
                        + (Az-As[j].z)*(Az-As[j].z));
            float db = sqrt((Bx-Bs[j].x)*(Bx-Bs[j].x)
                        + (By-Bs[j].y)*(By-Bs[j].y)
                        + (Bz-Bs[j].z)*(Bz-Bs[j].z));  
            tmp += (da-db) * (da-db);
        }
        sdata[tx] += tmp;
        __syncthreads();
    }
    
    for (unsigned int stride = blockDim.x/2; stride > 0; stride>>=1)
    {
        if (tx < stride)
        {
            sdata[tx] += sdata[tx + stride];
        }
        
        __syncthreads();
    }
    

    if (tx == 0) output[blockIdx.x] = sdata[0];
}


float solveGPU(sGalaxy A, sGalaxy B, int n) {
    float *hostOutput; 
    float *deviceOutput; 

    //determine correct number of output elements after reduction
    int numOutputElements = n / (blocksize / 2);
    if (n % (blocksize / 2)) {
        numOutputElements++;
    }

    hostOutput = (float *)malloc(numOutputElements * sizeof(float));
    // Round up according to array size 
    int gridSize = (n + blocksize - 1) / blocksize; 
    //printf("blocksize : %d gridSize: %d\n", blocksize, gridSize);
    //allocate GPU memory
    gpuSafeExec(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));
    //std::cerr << "galaxy_similarity_reduction<<<" << gridSize << "," << blocksize << "," << 0 << ">>>\n";
    galaxy_similarity_reduction<<<gridSize, blocksize>>>(A, B, n, deviceOutput);
    //move GPU results to CPU via PCIe
    gpuSafeExec(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

    //accumulate the sum in the first element
    for (int i = 1; i < numOutputElements; i++) {
        hostOutput[0] += hostOutput[i]; 
    }
    
    //use overall square root out of GPU, to avoid race condition
    float retval = sqrt(1/((float)n*((float)n-1)) * hostOutput[0]);

    //cleanup
    gpuSafeExec(hipFree(deviceOutput));
    free(hostOutput);

    return retval;
}