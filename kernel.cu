#include "hip/hip_runtime.h"
/*
    Ensures safe cuda application executions
*/
#define gpuSafeExec(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    Clears shared memory which is not full of previous
    numbers. Shmem is remembers values between consecutive
    kernel calls.
*/
__device__ void flushShmem(float *shmem, int shmemSize){
    for (int i = 0; i < shmemSize; i ++)
        shmem[i] = 0.0f;
    return;
}

struct sPoint{
    float x;
    float y;
    float z;
};

/*
Solved by 1d array reduction described by NVIDIA docs.
Might be improved with 2d array reduction?
*/
const int blocksize = 4;
__global__ void galaxy_similarity_reduction(const sGalaxy A, const sGalaxy B, int n , float* output) {
    __shared__ float sdata[blocksize];
    __shared__ sPoint As[blocksize];
    __shared__ sPoint Bs[blocksize];
    __shared__ sPoint Asj[blocksize];
    __shared__ sPoint Bsj[blocksize];

    unsigned int tid = threadIdx.x;
    unsigned int tx_g = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;

    As[tx].x = A.x[tx_g];
    As[tx].y = A.y[tx_g];
    As[tx].z = A.z[tx_g];
    Bs[tx].x = B.x[tx_g];
    Bs[tx].y = B.y[tx_g];
    Bs[tx].z = B.z[tx_g];

    //clear SHMEM
    if (tid == 0)
    {
        flushShmem(sdata, blocksize);
    }

    //wait for shem flush
    __syncthreads();

    for (int tile = blockIdx.x; tile < n / blocksize; tile++)
    {
        for (int j = 0; j < blocksize; j++)
        {
            Asj[tx].x = A.x[j + (blocksize * tile)];
            Asj[tx].y = A.y[j + (blocksize * tile)];
            Asj[tx].z = A.z[j + (blocksize * tile)];
            Bsj[tx].x = B.x[j + (blocksize * tile)];
            Bsj[tx].y = B.y[j + (blocksize * tile)];
            Bsj[tx].z = B.z[j + (blocksize * tile)];
            
        }
        __syncthreads();

            for (int j = 0; j < blocksize; j++){
                int idx = j + (blocksize * tile);
                if (idx > blocksize)
                {
                    printf("Loading required!\n");
                    //Bsj[tx].x = B.x[j + (blocksize * tile)];
                    //Bsj[tx].y = B.y[j + (blocksize * tile)];
                    //Bsj[tx].z = B.z[j + (blocksize * tile)];
                }
                
                if (idx < tx_g || idx == tx_g) continue;
                printf("idx: %d , j:%d \n", idx, j);
                
                float da = sqrt((As[tx].x-A.x[idx])*(As[tx].x-A.x[idx])
                            + (As[tx].y-A.y[idx])*(As[tx].y-A.y[idx])
                            + (As[tx].z-A.z[idx])*(As[tx].z-A.z[idx]));
                float db = sqrt((Bs[tx].x-B.x[idx])*(Bs[tx].x-B.x[idx])
                            + (Bs[tx].y-B.y[idx])*(Bs[tx].y-B.y[idx])
                            + (Bs[tx].z-B.z[idx])*(Bs[tx].z-B.z[idx]));
                /*
                float da = sqrt((As[tx].x-As[j].x)*(As[tx].x-As[j].x)
                            + (As[tx].y-As[j].y)*(As[tx].y-As[j].y)
                            + (As[tx].z-As[j].z)*(As[tx].z-As[j].z));
                float db = sqrt((Bs[tx].x-Bs[j].x)*(Bs[tx].x-Bs[j].x)
                            + (Bs[tx].y-Bs[j].y)*(Bs[tx].y-Bs[j].y)
                            + (Bs[tx].z-Bs[j].z)*(Bs[tx].z-Bs[j].z));
                */
                sdata[tx] += (da-db) * (da-db);
            }
        __syncthreads();
    }
    

    
    
    //do the math

    for(int j = tx_g + 1; j < n; j++){
        //printf("j: %d\n",j);
        //float da = sqrt((As[tx].x-A.x[j])*(As[tx].x-A.x[j])
        //            + (As[tx].y-A.y[j])*(As[tx].y-A.y[j])
        //            + (As[tx].z-A.z[j])*(As[tx].z-A.z[j]));
        //float db = sqrt((Bs[tx].x-B.x[j])*(Bs[tx].x-B.x[j])
        //            + (Bs[tx].y-B.y[j])*(Bs[tx].y-B.y[j])
        //            + (Bs[tx].z-B.z[j])*(Bs[tx].z-B.z[j]));
        //sdata[tid] += (da-db) * (da-db);
    }
    

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    if (tid == 0) output[blockIdx.x] = sdata[0];
}


float solveGPU(sGalaxy A, sGalaxy B, int n) {
    float *hostOutput; 
    float *deviceOutput; 

    //determine correct number of output elements after reduction
    int numOutputElements = n / (blocksize / 2);
    if (n % (blocksize / 2)) {
        numOutputElements++;
    }

    hostOutput = (float *)malloc(numOutputElements * sizeof(float));
    // Round up according to array size 
    int gridSize = (n + blocksize - 1) / blocksize; 
    //printf("blocksize : %d gridSize: %d\n", blocksize, gridSize);
    //allocate GPU memory
    gpuSafeExec(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));
    //std::cerr << "galaxy_similarity_reduction<<<" << gridSize << "," << blocksize << "," << 0 << ">>>\n";
    galaxy_similarity_reduction<<<gridSize, blocksize>>>(A, B, n, deviceOutput);
    //move GPU results to CPU via PCIe
    gpuSafeExec(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

    //accumulate the sum in the first element
    for (int i = 1; i < numOutputElements; i++) {
        hostOutput[0] += hostOutput[i]; 
    }
    
    //use overall square root out of GPU, to avoid race condition
    float retval = sqrt(1/((float)n*((float)n-1)) * hostOutput[0]);

    //cleanup
    gpuSafeExec(hipFree(deviceOutput));
    free(hostOutput);

    return retval;
}