#include "hip/hip_runtime.h"
/*
    Ensures safe cuda application executions
*/
#define gpuSafeExec(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    Clears shared memory which is not full of previous
    numbers. Shmem is remembers values between consecutive
    kernel calls.
*/
__device__ void flushShmem(float *shmem, int shmemSize){
    for (int i = 0; i < shmemSize; i ++)
        shmem[i] = 0.0f;
    return;
}


/*
Solved by 1d array reduction described by NVIDIA docs.
Might be improved with 2d array reduction?
*/
__global__ void galaxy_similarity_reduction(const sGalaxy A, const sGalaxy B, int n , float* output, int shmemSize) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    //clear SHMEM
    if (tid == 0)
    {
        flushShmem(sdata, shmemSize);

        for (int i = 0; i < shmemSize; i++)
        {
            if (sdata[i] != 0.0f)
            {
                printf("sdata[%d] = %f", i,sdata[i]);
            }  
        }
    }

    //wait for shem flush
    __syncthreads();
    
    //do the math
    for(int j = i+1; j < n; j++){
        float da = sqrt((A.x[i]-A.x[j])*(A.x[i]-A.x[j])
                    + (A.y[i]-A.y[j])*(A.y[i]-A.y[j])
                    + (A.z[i]-A.z[j])*(A.z[i]-A.z[j]));
        float db = sqrt((B.x[i]-B.x[j])*(B.x[i]-B.x[j])
                    + (B.y[i]-B.y[j])*(B.y[i]-B.y[j])
                    + (B.z[i]-B.z[j])*(B.z[i]-B.z[j]));
        sdata[tid] += (da-db) * (da-db);
    }

    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    if (tid == 0) output[blockIdx.x] = sdata[0];
}


float solveGPU(sGalaxy A, sGalaxy B, int n) {
    float *hostOutput; 
    float *deviceOutput; 

    int blockSize;
    int minGridSize;
    int gridSize;

    //use cuda occupancy calculator to determine grid and block sizes
    gpuSafeExec(hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
                                       galaxy_similarity_reduction, 0, 0)); 

    //determine correct number of output elements after reduction
    int numOutputElements = n / (blockSize / 2);
    if (n % (blockSize / 2)) {
        numOutputElements++;
    }

    hostOutput = (float *)malloc(numOutputElements * sizeof(float));
    // Round up according to array size 
    gridSize = (n + blockSize - 1) / blockSize; 

    //allocate GPU memory
    gpuSafeExec(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));

    galaxy_similarity_reduction <<<gridSize, blockSize, blockSize*sizeof(float) >>>(A, B, n, deviceOutput, blockSize);
    //move GPU results to CPU via PCIe
    gpuSafeExec(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

    //accumulate the sum in the first element
    for (int i = 1; i < numOutputElements; i++) {
        hostOutput[0] += hostOutput[i]; 
    }
    
    //use overall square root out of GPU, to avoid race condition
    float retval = sqrt(1/((float)n*((float)n-1)) * hostOutput[0]);

    //cleanup
    gpuSafeExec(hipFree(deviceOutput));
    free(hostOutput);

    return retval;
}