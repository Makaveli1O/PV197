#include "hip/hip_runtime.h"
/*
    Ensures safe cuda application executions
*/
#define gpuSafeExec(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    Clears shared memory which is not full of previous
    numbers. Shmem is remembers values between consecutive
    kernel calls.
*/
__device__ void flushShmem(float *shmem, int shmemSize){
    for (int i = 0; i < shmemSize; i ++)
        shmem[i] = 0.0f;
    return;
}

__device__ float diff(float Axi, float Axj, float Ayi, float Ayj, float Azi, float Azj
, float Bxi, float Bxj, float Byi, float Byj, float Bzi, float Bzj){
        float da = sqrt((Axi-Axj)*(Axi-Axj)
                    + (Ayi-Ayj)*(Ayi-Ayj)
                    + (Azi-Azj)*(Azi-Azj));
        float db = sqrt((Bxi-Bxj)*(Bxi-Bxj)
                    + (Byi-Byj)*(Byi-Byj)
                    + (Bzi-Bzj)*(Bzi-Bzj));
        return (da-db) * (da-db);
} 

const int blocksize = 256; //constant block size for higher Ns
/*
    Computation for higher numbers
*/
__global__ void galaxy_similarity_reduction(const sGalaxy A, const sGalaxy B, const int n , float* output) {
    __shared__ float sdata[blocksize];
    __shared__ float3 As[blocksize];
    __shared__ float3 Bs[blocksize];

    unsigned int tx_g = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;

    //clear SHMEM
    if (tx == 0)
    {
        flushShmem(sdata, blocksize);
    }

    //wait for shem flush
    __syncthreads();

    for (int tile = 0; tile < n / blocksize; tile++)
    {
        As[tx].x = A.x[tile*blocksize+tx];
        As[tx].y = A.y[tile*blocksize+tx];
        As[tx].z = A.z[tile*blocksize+tx];
        Bs[tx].x = B.x[tile*blocksize+tx];
        Bs[tx].y = B.y[tile*blocksize+tx];
        Bs[tx].z = B.z[tile*blocksize+tx];

        float Ax = A.x[tile*blocksize+tx];
        float Ay = A.y[tile*blocksize+tx];
        float Az = A.z[tile*blocksize+tx];
        float Bx = B.x[tile*blocksize+tx];
        float By = B.y[tile*blocksize+tx];
        float Bz = B.z[tile*blocksize+tx];
        float tmp = 0.0f;
        __syncthreads();
        for (int j = 1; j < blocksize; j++){
            int idx = j + (blocksize * tile); //global index   
            if (idx <= tx_g) continue;
            
            float da = sqrt((Ax-As[j].x)*(Ax-As[j].x)
                        + (Ay-As[j].y)*(Ay-As[j].y)
                        + (Az-As[j].z)*(Az-As[j].z));
            float db = sqrt((Bx-Bs[j].x)*(Bx-Bs[j].x)
                        + (By-Bs[j].y)*(By-Bs[j].y)
                        + (Bz-Bs[j].z)*(Bz-Bs[j].z));  
            tmp += (da-db) * (da-db);
        }
        sdata[tx] += tmp;
        __syncthreads();
    }
    
    for (unsigned int stride = blockDim.x/2; stride > 0; stride>>=1)
    {
        if (tx < stride)
        {
            sdata[tx] += sdata[tx + stride];
        }
        
        __syncthreads();
    }
    

    if (tx == 0) output[blockIdx.x] = sdata[0];
}
//algorithm for low N
__global__ void galaxy_similarity_reduction_lowN(const sGalaxy A, const sGalaxy B, const int n , float* output, const int blocksize) {
    extern __shared__ float sdata[];
    unsigned int tx_g = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;

    //clear SHMEM
    if (tx == 0)
    {
        flushShmem(sdata, blocksize);
    }

    //wait for shem flush
    __syncthreads();

    //load valus to registers
    float Ax = A.x[tx_g];
    float Ay = A.y[tx_g];
    float Az = A.z[tx_g];
    float Bx = B.x[tx_g];
    float By = B.y[tx_g];
    float Bz = B.z[tx_g];
    
    //do the math
    for(int j = tx_g+1; j < n; j++){
        float da = sqrt((Ax-A.x[j])*(Ax-A.x[j])
                    + (Ay-A.y[j])*(Ay-A.y[j])
                    + (Az-A.z[j])*(Az-A.z[j]));
        float db = sqrt((Bx-B.x[j])*(Bx-B.x[j])
                    + (By-B.y[j])*(By-B.y[j])
                    + (Bz-B.z[j])*(Bz-B.z[j]));
        sdata[tx] += (da-db) * (da-db);
    }

    for (unsigned int stride = blockDim.x/2; stride > 0; stride>>=1)
    {
        if (tx < stride)
        {
            sdata[tx] += sdata[tx + stride];
        }
        
        __syncthreads();
    }
    

    if (tx == 0) output[blockIdx.x] = sdata[0];

}

float solveGPU(sGalaxy A, sGalaxy B, int n) {
    float *hostOutput; 
    float *deviceOutput; 
    int _blocksize = blocksize;

    //determine block size
    if(n < _blocksize) _blocksize = n;

    //determine correct number of output elements after reduction
    int numOutputElements = n / (_blocksize / 2);
    if (n % (_blocksize / 2)) {
        numOutputElements++;
    }

    hostOutput = (float *)malloc(numOutputElements * sizeof(float));
    // Round up according to array size 
    int gridSize = (n + _blocksize - 1) / _blocksize; 
    //allocate GPU memory
    gpuSafeExec(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));
    if(n >= 3072){
        galaxy_similarity_reduction<<<gridSize, blocksize>>>(A, B, n, deviceOutput);
    }else{
        galaxy_similarity_reduction_lowN<<<gridSize, _blocksize, 2 * _blocksize * sizeof(float)>>>(A, B, n, deviceOutput, _blocksize);
    }
    //move GPU results to CPU via PCIe
    gpuSafeExec(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

    //accumulate the sum in the first element
    for (int i = 1; i < numOutputElements; i++) {
        hostOutput[0] += hostOutput[i]; 
    }
    
    //use overall square root out of GPU, to avoid race condition
    float retval = sqrt(1/((float)n*((float)n-1)) * hostOutput[0]);

    //cleanup
    gpuSafeExec(hipFree(deviceOutput));
    free(hostOutput);

    return retval;
}
