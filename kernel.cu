#include "hip/hip_runtime.h"
//TODO kernel implementation

/*
Performs iteration paraller calculation no GPU
*/

#define NUM_BLOCKS = 10
#define NUM_THREADS = 256

__global__ void increment_gpu(sGalaxy A, sGalaxy B, int n, float* result){
    //determine unique index within grid, miximizing block usage
    __shared__ float d_result;
    d_result = 0.0f;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = i + 1;
    //use odd/even numbers to i/j index?
    //FIXME wrong arithmetic. Calculations does not work. Probably stepping out array?
    float da = sqrt((A.x[i]-A.x[j])*(A.x[i]-A.x[j])
				+ (A.y[i]-A.y[j])*(A.y[i]-A.y[j])
				+ (A.z[i]-A.z[j])*(A.z[i]-A.z[j]));
    float db = sqrt((B.x[i]-B.x[j])*(B.x[i]-B.x[j])
				+ (B.y[i]-B.y[j])*(B.y[i]-B.y[j])
				+ (B.z[i]-B.z[j])*(B.z[i]-B.z[j]));
    float tmp = (da-db) * (da-db);

    d_result += tmp;
    *result = sqrt(1/((float)n*((float)n-1)) * d_result);
    printf("Inside: %f \n", *result);
    return;

}


float solveGPU(sGalaxy A, sGalaxy B, int n) {
    float h_result[n];
    float *d_result;

    //allocate space on GPU for d_result variable
    hipMalloc((void**)&d_result, sizeof(float));

    //define grid and block sizes
    dim3 grid_size(1);
    dim3 block_size(3);

    //Launch kernel with pass by referennce attribute and N size
    increment_gpu <<<grid_size, block_size>>>(A, B, n, d_result);

    //copy data from GPU memory to CPU memory via PCIe bus
    hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    return *h_result;
}